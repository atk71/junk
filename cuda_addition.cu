#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define N 10000000

__global__ void vector_add(int *c, int *a, int *b, int n) {
	for (int i = 0; i < n; i++) {
		c[i] = a[i] + b[i];
	}
}
int main() {
	int* a1;
	int* b1;
	int* c1;
	int* a2;
	int* b2;
	int* c2;
	
	a1 = (int*) malloc((sizeof(int) * N));
	b1 = (int*) malloc((sizeof(int) * N));
	c1 = (int*) malloc((sizeof(int) * N));
	
	hipMalloc((void**)&a2, sizeof(int) * N);
	hipMalloc((void**)&b2, sizeof(int) * N);
	hipMalloc((void**)&c2, sizeof(int) * N);
	
	for (int i = 0; i < N; i++) {
		a[i] = 0;
		b[i] = 1;
	}
	
	hipMemcpy(a2, a1, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(b2, b1, sizeof(int) * N, hipMemcpyHostToDevice);
	
	vector_add<<<1, 1>>>(c2, a2, b2, N);
	
	hipMemcpy(c1, c2, sizeof(int) * N, hipMemcpyHostToDevice);
	
	hipFree(a2);
	hipFree(b2);
	hipFree(c2);
	
	free(a1);
	free(b1);
	free(c1);
	
	return 0;
}